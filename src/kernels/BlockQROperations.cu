#include "hip/hip_runtime.h"
#include "../BlockQROperations.h"
#include "../constants.h"
#include "../error.h"
#include "../Vector.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

/**
  * Given an n-vector x, computes an n-vector v with v[0] = 1 such that (I - 2*v*t(v) / t(v) * v)x is 
  * zero in all but the first component.
  * @param x n-vector used to compute v.
  * @param v n-vector to store output
  * @param n Length of vectors x and v
  *
  * @see{https://www.youtube.com/watch?v=d-yPM-bxREs}
  */
__device__ void house(hipblasHandle_t *handle, Numeric *x, Numeric *v, int n)
{
    Numeric x_norm;

    #if FLOAT_NUMERIC
    	hipblasScopy(*handle, n, x, 1, v, 1);
    	hipblasSnrm2(*handle, n, x, 1, &x_norm);
    #else
    	hipblasDcopy(*handle, n, x, 1, v, 1);
    	hipblasDnrm2(*handle, n, x, 1, &x_norm);
    #endif

    if (x_norm != 0) {
    	const Numeric sign = x[0] >= 0 ? 1.0 : -1.0;
    	const Numeric beta = 1.0 / (x[0] + (sign * x_norm));
    	#if FLOAT_NUMERIC
    		hipblasSscal(*handle, n - 1, &beta, &v[1], 1);
    	#else
    		hipblasDscal(*handle, n - 1, &beta, &v[1], 1);
    	#endif
    }
    v[0] = 1.0;
}

__global__ void Block_house_kernel(hipblasHandle_t *handle, Numeric *x, Numeric *v, int n) {
    house(handle, x, v, n);
}

extern "C"
int
Block_house(hipblasHandle_t *handle, Vector *in, Vector *out) {
    dim3 dimGrid(1, 1);
    dim3 dimBlock(1, 1);
    
    Block_house_kernel<<<dimGrid, dimBlock>>>(handle, in->data_d, out->data_d, in->nr_elems);
    return 0;
}