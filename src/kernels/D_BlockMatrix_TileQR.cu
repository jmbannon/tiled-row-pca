#include "hip/hip_runtime.h"
#include "../TileQR_Operations.h"
#include "../constants.h"
#include "../error.h"
#include "../Vector.h"
#include "../Matrix.h"
#include "../BlockMatrix.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>
#include <stdbool.h>

////////////////////////////////////////////////////////////////////////////////////////////////////
// DGEQT2
////////////////////////////////////////////////////////////////////////////////////////////////////

/**
  * Given an n-vector x, computes an n-vector v with v[0] = 1 such that (I - 2*v*t(v) / t(v) * v)x is 
  * zero in all but the first component.
  * @param x n-vector used to compute v.
  * @param v n-vector to store output
  * @param n Length of vectors x and v
  *
  * @see{https://www.youtube.com/watch?v=d-yPM-bxREs}
  */
__device__ int house(hipblasHandle_t *handle, Numeric *x, Numeric *v, int n)
{
    int res;
    Numeric x_norm;

    // Copies x into v and calculates the norm
    #if FLOAT_NUMERIC
    	res = hipblasScopy(*handle, n, x, 1, v, 1);
      CHECK_CUBLAS_RETURN(res, "Failed to copy vector x");

    	res = hipblasSnrm2(*handle, n, x, 1, &x_norm);
    #else
    	res = hipblasDcopy(*handle, n, x, 1, v, 1);
      CHECK_CUBLAS_RETURN(res, "Failed to copy vector x");

    	res = hipblasDnrm2(*handle, n, x, 1, &x_norm);
    #endif
    CHECK_CUBLAS_RETURN(res, "Failed to calculate norm of x");

    if (x_norm != 0) {
    	const Numeric sign = x[0] >= 0 ? 1.0 : -1.0;
    	const Numeric beta = 1.0 / (x[0] + (sign * x_norm));
    	#if FLOAT_NUMERIC
    		res = hipblasSscal(*handle, n - 1, &beta, &v[1], 1);
    	#else
    		res = hipblasDscal(*handle, n - 1, &beta, &v[1], 1);
    	#endif
      CHECK_CUBLAS_RETURN(res, "Failed scale vector v");
    }
    v[0] = 1.0;

    return res;
}

/**
  * Overwrites A with PA where P = (I - 2*v*t(v) / t(v) * v).
  * The following algorithm is from Golub, Van Loan Matrix Computations:
  *
  * function: A = row.house(A, v)
  *     beta = -2 / (t(v) * v)
  *     w = beta * t(A) * v
  *     A = A + v * t(w)
  *
  * @param A m-by-n matrix
  * @param v m-vector v with v[0] = 1.0 {@see house}
  * @param beta Scalar used in the transformation.
  * @param w n-vector temporary storage
  */
__device__ int house_row(hipblasHandle_t *handle, Numeric *A, Numeric *v, Numeric *beta, Numeric *w, int m, int n, int ldm)
{
    int res;

    // Computes beta
    #if FLOAT_NUMERIC
      res = hipblasSnrm2(*handle, m, v, 1, beta);
    #else
      res = hipblasDnrm2(*handle, m, v, 1, beta);
    #endif
    CHECK_CUBLAS_RETURN(res, "Failed to compute beta");

    *beta = -2.0 / (*beta * *beta);

    // Computes w
    Numeric w_scalar = 0.0;
    #if FLOAT_NUMERIC
      res = hipblasSgemv(*handle, HIPBLAS_OP_T, m, n, beta, A, ldm, v, 1, &w_scalar, w, 1);
    #else
      res = hipblasDgemv(*handle, HIPBLAS_OP_T, m, n, beta, A, ldm, v, 1, &w_scalar, w, 1);
    #endif
    CHECK_CUBLAS_RETURN(res, "Failed to compute w");

    // Annihilate column of A: A = A + v * t(w)
    Numeric scalar = 1.0;
    #if FLOAT_NUMERIC
      res = hipblasSgemm(*handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, 1, &scalar, v, m, w, 1, &scalar, A, ldm);
    #else
      res = hipblasDgemm(*handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, 1, &scalar, v, m, w, 1, &scalar, A, ldm);
    #endif
    CHECK_CUBLAS_RETURN(res, "Failed to annihilate column in A");

    return res;
}

/**
  * Produces an upper triangular matrix R, unit lower triangular matrix V that contains n Householder reflectors.
  * R and V are written on the memory area used for A.
  *
  * @param A upper triangular matrix R. Lower triangular contains partial householder reflectors:
  *        all diagonal elements should be 1 to represent full householder reflector.
  * @param w Work-space vector.
  * @param store_house True if householder vectors should be stored in lower-triangular portion of output. False otherwise.
  */
__device__ int house_qr(hipblasHandle_t *handle, Numeric *A, Numeric *beta, Numeric *w, bool store_house, int m, int n)
{
  int res;
  Numeric *v;

  res = hipMalloc(&v, m * sizeof(Numeric));
  CHECK_SUCCESS_RETURN(res);

  for (int j = 0; j < n; j++) {
    int pos = MAT_POS(j, j, m);
    res = house(handle, &A[pos], &v[j], m - j);
    CHECK_ZERO_ERROR_RETURN(res, "Failed to compute house");

    res = house_row(handle, &A[pos], &v[j], &beta[j], &w[j], m - j, n - j, m);
    CHECK_ZERO_ERROR_RETURN(res, "Failed to compute house_row");

    // Copies householder vector into lower triangular portion of A
    if (store_house && j < m) {
      #if FLOAT_NUMERIC
        res = hipblasScopy(*handle, m - j - 1, &v[j + 1], 1, &A[pos + 1], 1);
      #else
        res = hipblasDcopy(*handle, m - j - 1, &v[j + 1], 1, &A[pos + 1], 1);
      #endif
      CHECK_CUBLAS_RETURN(res, "Failed to copy householder vector into lower-triangular portion of A");
    }
  }

  res = hipFree(v);
  CHECK_SUCCESS_RETURN(res);

  return res;
}


/**
  * Computes the Matrix T such that P_1 * ... * P_n = I + (Y * T * t(Y))
  * where P_i are the householder matrices that upper triangularize A (i.e. R in QR) during the ith step.
  * 
  * For j = 1:r
  *   if j == 1 then
  *     Y = [v_1]; T = [-2]
  *   else
  *     z = -2 * T * t(Y) * v_j
  *     Y = [Y v_j]
  *
  *     T = [T  z]
  *         [0 -2]
  *   endif
  * end j
  *
  * Note it assumes householder vectors in Y are normalized. We change the algorithm slightly to use 
  * previously calculated betas (-2 / t(v) * v == -2 / ||v||^2). We replace '-2' with beta[j].
  *
  * @see{https://www.cs.cornell.edu/cv/ResearchPDF/A%20Storage-Efficient%20WY%20Representation%20for%20Products%20of%20Householder%20Transformations.pdf}
  *
  * @param Y m-by-n matrix where lower-triangular + diag portion holds householder vectors and the upper-triangular portion holds the R matrix from QR.
  * @param T n-by-n output matrix to store T
  */
__device__ int house_yt(hipblasHandle_t *handle, Numeric *Y, Numeric *T, Numeric *beta, int m, int n)
{
  int res;
  Numeric alpha;
  Numeric zero = 0.0;
  int v_idx;
  int z_idx;
  int y_idx;

  T[0] = beta[0];
  for (int j = 1; j < n; j++) {
    alpha = beta[j];

    y_idx = MAT_POS(j, 0, m);
    v_idx = MAT_POS(j, j, m);
    z_idx = MAT_POS(0, j, n);

    // Computes -2 * t(Y) * v_j = z' in an optimized way to ignore 0 elements in v_j. Stores it in z-location of T matrix.
    #if FLOAT_NUMERIC
      res = hipblasSgemm(*handle, HIPBLAS_OP_T, HIPBLAS_OP_N, j, 1, m - j, &alpha, &Y[y_idx], m, &Y[v_idx], m, &zero, &T[z_idx], n);
    #else
      res = hipblasDgemm(*handle, HIPBLAS_OP_T, HIPBLAS_OP_N, j, 1, m - j, &alpha, &Y[y_idx], m, &Y[v_idx], m, &zero, &T[z_idx], n);
    #endif
    CHECK_CUBLAS_RETURN(res, "Failed to compute -2 * t(Y) * v_j matrix-matrix multiplication in house_yt");

    // Computes T * z' using a triangular matrix-vector multiplication routine.
    #if FLOAT_NUMERIC
      res = hipblasStrmv(*handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, j, T, n, &T[z_idx], 1);
    #else
      res = hipblasDtrmv(*handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, j, T, n, &T[z_idx], 1);
    #endif
    CHECK_CUBLAS_RETURN(res, "Failed to compute T * z' triangular matrix-vector multiplication in house_yt");

    T[MAT_POS(j, j, n)] = beta[j];
  }
  return 0;
}

/**
  * Performs QR decomposition on a m-by-n matrix A. Computes upper-triangular matrix R,
  * where A = QR and Householder vectors Y are stored in lower-diagonal portion of R.
  * Uses Y to compute T, where Q = I + Y %*% T %*% t(Y).
  *
  * @param A m-by-n matrix.
  * @param T n-by-n output matrix.
  * @return R, Y, T, where A = QR and T for Q = I + Y %*% T %*% t(Y). Overwrites A with R and Y.
  */
__device__ int dgeqt2(hipblasHandle_t *handle, Numeric *A, Numeric *T, int m, int n)
{
  int res;
  // Temporary work matrix
  Numeric *w;
  Numeric *beta;

  res = hipMalloc(&w, m * sizeof(Numeric));
  CHECK_SUCCESS_RETURN(res);

  res = hipMalloc(&beta, n * sizeof(Numeric));
  CHECK_SUCCESS_RETURN(res);

  res = house_qr(handle, A, beta, w, true, m, n);
  CHECK_ZERO_ERROR_RETURN(res, "Failed to compute house_qr in dgeqt2");

  // Restore householder vectors for YT Generation. Store diag in work vector.
  int diag_idx;
  for (int i = 0; i < n; i++) {
    diag_idx = MAT_POS(i, i, m);
    w[i] = A[diag_idx];
    A[diag_idx] = 1.0;

  }

  res = house_yt(handle, A, T, beta, m, n);
  CHECK_ZERO_ERROR_RETURN(res, "Failed to compute house_yt in dgeqt2");

  for (int i = 0; i < n; i++) {
    diag_idx = MAT_POS(i, i, m);
    A[diag_idx] = w[i];
  }

  res = hipFree(&w);
  CHECK_SUCCESS_RETURN(res);

  res = hipFree(&beta);
  CHECK_SUCCESS_RETURN(res);

  return 0;
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// DTSQT2
////////////////////////////////////////////////////////////////////////////////////////////////////

/**
  * Performs DGEQT2 on a row-binded matrix rbind(R, A).
  *
  * @param handle cuBLAS handle
  * @param R n-by-n upper-triangular matrix to row-bind. May have non-zero elements in lower diagonal. Stores output R matrix from DGEQT2 here.
  * @param A n-by-n matrix to row-bind. Stores lower portion of householder vectors from DGEQT2 in here. The 'hessianberg' portion of the householder
  *          vectors is an identity matrix, so there is no need to store that.
  * @param T n-by-n output matrix.
  * @param RA_rowbind 2n-by-n work matrix to store the row-bind and compute DGEQT2 on.
  * @param zero_tri True if lower-triangular portion of R needs to be zeroed. False otherwise.
  */
__device__ int dtsqt2(hipblasHandle_t *handle, Numeric *R, Numeric *A, Numeric *T, Numeric *RA_rowbind, bool zero_tri, int n)
{
  int res;
  int RArows = 2*n;

  // TODO: Optimize

  // Stores R into upper-portion of RA_rowbind. Zeroes lower-triangular portion.
  for (int j = 0; j < n; j++) {
    for (int i = 0; i < n; i++) {
      RA_rowbind[MAT_POS(i, j, RArows)] = (i > j) ? 0.0 : R[MAT_POS(i, j, n)];
    }
  }

  // Stores A into lower-portion of RA_rowbind.
  for (int j = 0; j < n; j++) {
    for (int i = 0; i < n; i++) {
      RA_rowbind[MAT_POS(n + i, j, RArows)] = A[MAT_POS(i, j, n)];
    }
  }

  res = dgeqt2(handle, RA_rowbind, T, RArows, n);
  CHECK_ZERO_ERROR_RETURN(res, "Failed to compute dgeqt2 on row-binded matrix");

  // Stores output R matrix into upper-triangular portion of R
  for (int j = 0; j < n; j++) {
    for (int i = 0; i <= j; i++) {
      R[MAT_POS(i, j, n)] = RA_rowbind[MAT_POS(i, j, RArows)];
    }
  }

  // Stores output householder vectors into A.
  for (int j = 0; j < n; j++) {
    for (int i = 0; i < n; i++) {
      A[MAT_POS(i, j, n)] = RA_rowbind[MAT_POS(n + i, j, RArows)];
    }
  }

  return 0;
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// DSSRFB
////////////////////////////////////////////////////////////////////////////////////////////////////

/**
  * Performs DSSRFB
  * The original operation is rbind(A_kj, A_ij) = t(I + (V * T * t(V))) * rbind(A_kj, A_ij)
  * It can be optimized to the following:
  * Let X = t(T) * A_kj
  * Let Y = t(T) * t(V) * A_ij
  * Let Z = X + Y
  *
  * rbind(A_kj, A_ij) = rbind(A_kj + Z, A_ij + (V * Z))
  *
  *
  *
  */
__device__ int dssrfb(hipblasHandle_t *handle,
                      Numeric *A_kj,
                      Numeric *A_ij,
                      Numeric *V,
                      Numeric *T,
                      Numeric *X, int ldx,
                      Numeric *Y, int ldy,
                      int n)
{
  int res;
  Numeric alpha = 1.0;
  Numeric zero = 0.0;
  #if FLOAT_NUMERIC
    res = hipblasSgemm(*handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, n, n, &alpha, T, n, A_kj, n, &zero, X, ldx);
  #else
    res = hipblasDgemm(*handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, n, n, &alpha, T, n, A_kj, n, &zero, X, ldx);
  #endif
  CHECK_CUBLAS_RETURN(res, "Failed to compute X = t(T) * A_kj");

  #if FLOAT_NUMERIC
    res = hipblasSgemm(*handle, HIPBLAS_OP_T, HIPBLAS_OP_T, n, n, n, &alpha, T, n, V, n, &zero, Y, ldy);
  #else
    res = hipblasDgemm(*handle, HIPBLAS_OP_T, HIPBLAS_OP_T, n, n, n, &alpha, T, n, V, n, &zero, Y, ldy);
  #endif
  CHECK_CUBLAS_RETURN(res, "Failed to compute Y' = t(T) * t(V)");

  hipDeviceSynchronize();

  #if FLOAT_NUMERIC
    res = hipblasSgemm(*handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, Y, ldy, A_ij, n, &alpha, X, ldx);
  #else
    res = hipblasDgemm(*handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, Y, ldy, A_ij, n, &alpha, X, ldx);
  #endif
  CHECK_CUBLAS_RETURN(res, "Failed to compute Z = X = Y' * A_ij + X");

  #if FLOAT_NUMERIC
    res = hipblasSgeam(*handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, &alpha, A_kj, n, &alpha, X, ldx, A_kj, n);
  #else
    res = hipblasDgeam(*handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, &alpha, A_kj, n, &alpha, X, ldx, A_kj, n);
  #endif
  CHECK_CUBLAS_RETURN(res, "Failed to compute A_kj = A_kj + Z");

  #if FLOAT_NUMERIC
    res = hipblasSgemm(*handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, V, n, X, ldx, &alpha, A_ij, n);
  #else
    res = hipblasDgemm(*handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, V, n, X, ldx, &alpha, A_ij, n);
  #endif
  CHECK_CUBLAS_RETURN(res, "Failed to compute A_ij = (V * Z) + A_ij");

  return res;
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// DLAFRB
////////////////////////////////////////////////////////////////////////////////////////////////////


/**
 * Multiplies a m-by-k Hessianberg matrix with a k-by-n matrix.
 * Multiplies its triangle portion using optimized triangle-matrix multiply.
 *
 * C = alpha * H * B
 *
 * @param handle cuBLAS handle
 * @param diag Whether the diagonal in the Hessian is Unit (1's) or Non-Unit.
 * @param m Rows of Hessianberg matrix H and C
 * @param n Cols of B and C
 * @param k Cols of Hessianberg matrix H and rows of B
 * @param alpha Scalar
 * @param A m-by-k Hessianberg matrix
 * @param B k-by-n Matrix
 * @param C m-by-n Matrix
 */
__device__ int cublasDgemm_hmn(hipblasHandle_t handle,
                               hipblasDiagType_t diag,
                               int m, int n, int k,
                               const Numeric alpha,
                               const Numeric *A, int lda,
                               const Numeric *B, int ldb,
                               Numeric *C, int ldc)
{

  int res;
  Numeric zero = 0.0;
  // Multiply the triangular portion
  #if FLOAT_NUMERIC
    res = hipblasStrmm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, diag, k, n, &alpha, A, lda, B, ldb, C, ldc);
  #else
    res = hipblasDtrmm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, diag, k, n, &alpha, A, lda, B, ldb, C, ldc);
  #endif 
  CHECK_CUBLAS_RETURN(res, "Triangle portion of Hessianberg matrix multiply failed");

  if (m != k) {
    // Multiply the rectangular portion if the Hessianberg matrix H is not a triangular matrix
    #if FLOAT_NUMERIC
      res = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m - k, n, k, &alpha, &A[k], lda, B, ldb, &zero, &C[k], ldc);
    #else
      res = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m - k, n, k, &alpha, &A[k], lda, B, ldb, &zero, &C[k], ldc);
    #endif
    CHECK_CUBLAS_RETURN(res, "Rectangle portion of Hessianberg matrix multiply failed");
  }
  return 0;
}

/**
 * Multiplies a m-by-k matrix with a n-by-k transposed Hessianberg matrix.
 * Multiplies its triangle portion using optimized triangle-matrix multiply.
 *
 * C = alpha * A * t(H)
 *
 * @param handle cuBLAS handle
 * @param diag Whether the diagonal in the Hessian is Unit (1's) or Non-Unit.
 * @param m Rows of matrix A and C
 * @param n Cols of t(H) and C (rows of H)
 * @param k Cols of A and rows of t(H) (or cols of H)
 * @param alpha Scalar
 * @param A m-by-k Matrix
 * @param H n-by-k Hessianberg matrix to be multiplied transposed
 * @param C m-by-n Matrix
 */
__device__ int cublasDgemm_mht(hipblasHandle_t handle,
                               hipblasDiagType_t diag,
                               int m, int n, int k,
                               const Numeric alpha,
                               const Numeric *A, int lda,
                               const Numeric *B, int ldb,
                               Numeric *C, int ldc)
{
  int res;
  Numeric zero = 0.0;
  // Multiply the triangular portion
  #if FLOAT_NUMERIC
    res = hipblasStrmm(handle, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_T, diag, m, k, &alpha, B, ldb, A, lda, C, ldc);
  #else
    res = hipblasDtrmm(handle, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_T, diag, m, k, &alpha, B, ldb, A, lda, C, ldc);
  #endif 
  CHECK_CUBLAS_RETURN(res, "Triangle portion of Hessianberg matrix multiply failed");
  hipDeviceSynchronize();

  if (n != k) {
    // Multiply the rectangular portion if the Hessianberg matrix H is not a triangular matrix
    int C_rectangle_idx = MAT_POS(0, k, ldc);
    #if FLOAT_NUMERIC
      res = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, k, &alpha, A, lda, &B[k], ldb, &zero, &C[C_rectangle_idx], ldc);
    #else
      res = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, k, &alpha, A, lda, &B[k], ldb, &zero, &C[C_rectangle_idx], ldc);
    #endif
    CHECK_CUBLAS_RETURN(res, "Rectangle portion of Hessianberg matrix multiply failed");
    hipDeviceSynchronize();
  }

  return 0;
}


/**
  * Computes Q = I + (Y * T * t(Y))
  *
  * @param handle cuBLAS handle
  * @param Y m-by-n Hessianberg matrix containing householder vectors in lower portion.
  * @param T n-by-n Upper-triangular matrix
  * @param Q_ m-by-n work matrix to store Y * T
  * @return Q m-by-m output matrix
  */
__device__ int house_qr_q(hipblasHandle_t *handle, Numeric *Y, Numeric *T, Numeric *Q, Numeric *Q_, int m, int n)
{
    int res;
    Numeric alpha = 1.0;
    
    // Calculates Q' = Y * T
    res = cublasDgemm_hmn(*handle, HIPBLAS_DIAG_UNIT, m, n, n, alpha, Y, m, T, n, Q_, m);
    CHECK_CUBLAS_RETURN(res, "Failed to compute Q' = Y * T");
    hipDeviceSynchronize();

    // Calculates Q = Q' * t(Y)
    //              = Y * T * t(Y)
    res = cublasDgemm_mht(*handle, HIPBLAS_DIAG_UNIT, m, m, n, alpha, Q_, m, Y, m, Q, m);
    CHECK_CUBLAS_RETURN(res, "Failed to compute Q = Q' * t(Y)");
    hipDeviceSynchronize();

    // Calculates Q = I + Q
    //              = I + (Y * T * t(Y))
    for (int i = 0; i < m; i++) {
      Q[MAT_POS(i, i, m)] += 1.0;
    }

    return 0;
}


/**
  * Multiplies a matrix A s.t. A = t(Q) * A
  *                              = t(I + (Y * T * t(Y))) * A
  * where Q is from a diagonal tile P, where P = QR, and A is an adjacent tile to the right of P.
  *
  * @param handle cuBLAS handle
  * @param A m-by-n matrix to multiply and override. Adjacent to the source tile of Q.
  * @param Y m-by-n Hessianberg matrix holding householder vectors.
  * @param T n-by-n matrix.
  * @param Q m-by-m matrix to store Q.
  * @param Q_ m-by-n work matrix.
  */
__device__ int dlarfb(hipblasHandle_t *handle, Numeric *A, Numeric *Y, Numeric *T, Numeric *Q, Numeric *Q_, int m, int n)
{
  int res;

  res = house_qr_q(handle, Y, T, Q, Q_, m, n);
  CHECK_ZERO_ERROR_RETURN(res, "Failed to compute house_qr_q");

  Numeric zero = 0.0;
  Numeric alpha = 1.0;
  #if FLOAT_NUMERIC
    res = hipblasSgemm(*handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, m, &alpha, Q, m, A, m, &zero, Q_, m);
  #else
    res = hipblasDgemm(*handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, m, &alpha, Q, m, A, m, &zero, Q_, m);
  #endif
  CHECK_CUBLAS_RETURN(res, "Failed to compute Q' = t(Q) * A")


  #if FLOAT_NUMERIC
    res = hipblasScopy(*handle, m * n, Q_, 1, A, 1);
  #else
    res = hipblasDcopy(*handle, m * n, Q_, 1, A, 1);
  #endif
  CHECK_CUBLAS_RETURN(res, "Failed to copy A = Q'")

  return 0;
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// TileQR
////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ int BlockMatrix_TileQR_single_thread_kernel(Numeric *A, int blk_m, int blk_n)
{
  int res;
  Numeric *T;
  Numeric *Rbind;
  Numeric *Q;
  Numeric *Q_;

  int min_blk_d = blk_m > blk_n ? blk_n : blk_m;

  hipblasHandle_t handle;
  res = hipblasCreate(&handle);
  CHECK_CUBLAS_RETURN(res, "Failed to init handle");

  res = hipMalloc(&T, BLK_SIZE_MEM);
  CHECK_CUBLAS_RETURN(res, "Failed to init T");

  res = hipMalloc(&Rbind, 2 * BLK_SIZE_MEM);
  CHECK_CUBLAS_RETURN(res, "Failed to init Rbind");

  res = hipMalloc(&Q, BLK_SIZE_MEM);
  CHECK_CUBLAS_RETURN(res, "Failed to init Q");

  res = hipMalloc(&Q_, BLK_SIZE_MEM);
  CHECK_CUBLAS_RETURN(res, "Failed to init Q'");

  for (int i = 0; i < BLK_SIZE; i++) {
    T[i] = 0;
    Q[i] = 0;
    Q_[i] = 0;
    Rbind[i] = 0;    
  }

  for (int i = BLK_SIZE; i < 2*BLK_SIZE; i++) {
    Rbind[i] = 0;
  }

  for (int k = 0; k < min_blk_d; k++) {
    Numeric *A_kk = &A[BLK_POS(k, k, blk_n)];

    res = dgeqt2(&handle, A_kk, T, BLK_LEN, BLK_LEN);
    CHECK_ZERO_ERROR_RETURN(res, "Failed to compute dgeqt2");

    for (int n = (k + 1); n < blk_n; n++) {

      Numeric *A_kn = &A[BLK_POS(k, n, blk_n)];

      res = dlarfb(&handle, A_kn, A_kk, T, Q, Q_, BLK_LEN, BLK_LEN);
      CHECK_ZERO_ERROR_RETURN(res, "Failed to compute dlarfb");

    }

    for (int m = (k + 1); m < blk_m; m++) {

      Numeric *A_mk = &A[BLK_POS(m, k, blk_n)];
      for (int i = 0; i < BLK_SIZE; i++) {
        T[i] = 0;  
      }

      res = dtsqt2(&handle, A_kk, A_mk, T, Rbind, true, BLK_LEN);
      CHECK_ZERO_ERROR_RETURN(res, "Failed to compute dtsqt2");

      for (int n = (k + 1); n < blk_n; n++) {
        Numeric *A_kn = &A[BLK_POS(k, n, blk_n)];
        Numeric *A_mn = &A[BLK_POS(m, n, blk_n)];

        res = dssrfb(&handle, A_kn, A_mn, A_mk, T, Rbind, DBL_BLK_LEN, &Rbind[BLK_LEN], DBL_BLK_LEN, BLK_LEN);
        CHECK_ZERO_ERROR_RETURN(res, "Failed to compute dssrfb");
      }
    }
  }

  res = hipFree(&T);
  CHECK_SUCCESS_RETURN(res);

  res = hipFree(&Rbind);
  CHECK_SUCCESS_RETURN(res);

  res = hipFree(&Q);
  CHECK_SUCCESS_RETURN(res);

  res = hipFree(&Q_);
  CHECK_SUCCESS_RETURN(res);

  return 0;
}

//////////////////////
// Parallel QR
//////////////////////


/**
 * Performs DGEQT2 on a diagonal block of the BlockMatrix A.
 *
 * @param A Pointer to start of A
 * @param ldba Leading block dimension of A
 * @param T BLK_LEN-by-BLK_LEN storage matrix for result
 * @param k Block row and column (diagonal block) to perform DGEQT2 on
 */
__global__ void dgeqt2_kernel(Numeric *A, Numeric *T) {
    hipblasHandle_t handle;
    int res = hipblasCreate(&handle);

    dgeqt2(&handle, A, T, BLK_LEN, BLK_LEN);
}

/**
  * Multiplies a matrix A s.t. A = t(Q) * A
  *                              = t(I + (Y * T * t(Y))) * A
  * where Q is from a diagonal tile P, where P = QR, and A is an adjacent tile to the right of P.
  *
  * @param M Pointer to beginning of BlockMatrix M
  * @param i Block row to perform on
  * @param T n-by-n matrix
  *
  */

  /**
  * Multiplies a matrix A s.t. A = t(Q) * A
  *                              = t(I + (Y * T * t(Y))) * A
  * where Q is from a diagonal tile P, where P = QR, and A is an adjacent tile to the right of P.
  *
  * @param handle cuBLAS handle
  * @param A m-by-n matrix to multiply and override. Adjacent to the source tile of Q.
  * @param Y m-by-n Hessianberg matrix holding householder vectors.
  * @param T n-by-n matrix.
  * @param Q m-by-m matrix to store Q.
  * @param Q_ m-by-n work matrix.
  *
  * res = dlarfb(&handle, A_kn, A_kk, T, Q, Q_, BLK_LEN, BLK_LEN);
  */
__global__ void dlarfb_kernel(Numeric *M, int lbdm, int k, Numeric *T) {
    Numeric Q[BLK_SIZE];
    Numeric Q_[BLK_SIZE];
    hipblasHandle_t handle;
    int res = hipblasCreate(&handle);
    // check res

    Numeric *M_kk = &M[BLK_POS(k, k, lbdm)];
    Numeric *M_kn = &M[BLK_POS(k, k + 1 + threadIdx.x, lbdm)];

    res = dlarfb(&handle, M_kn, M_kk, T, Q, Q_, BLK_LEN, BLK_LEN);
    // check res
}

__global__ void dtsqt2_dssrfb_row_kernel(Numeric *M, int lbdm, int k, int m) {
  hipblasHandle_t handle;
  int res = hipblasCreate(&handle);
  // check res

  __shared__ Numeric T[BLK_SIZE];
  __shared__ Numeric Rbind[DBL_BLK_SIZE];
  __shared__ bool mutex;

  if (threadIdx.x == 0) {
    mutex = true;
  }
  __syncthreads();

  Numeric *A_mk = &M[BLK_POS(m, k, lbdm)];
  Numeric *A_kn = &M[BLK_POS(k, k + 1 + threadIdx.x, lbdm)];
  Numeric *A_mn = &M[BLK_POS(m, k + 1 + threadIdx.x, lbdm)];

  if (threadIdx.x == 0) {
    Numeric *A_kk = &M[BLK_POS(k, k, lbdm)];

    res = dtsqt2(&handle, A_kk, A_mk, T, Rbind, true, BLK_LEN); // check res
    //CHECK_ZERO_ERROR_RETURN(res, "Failed to compute dtsqt2");

    mutex = false;
  } else {
    while(mutex) {}
  }

  res = dssrfb(&handle, A_kn, A_mn, A_mk, T, Rbind, DBL_BLK_LEN, &Rbind[BLK_LEN], DBL_BLK_LEN, BLK_LEN); // check res
  //CHECK_ZERO_ERROR_RETURN(res, "Failed to compute dssrfb");
}

extern "C"
int
BlockMatrix_TileQR_multi_thread(BlockMatrix *BlkM)
{    
  int res;
  Numeric *M;
  Numeric *T;

  M = BlkM->data;
  int blk_m = BlkM->nr_blk_rows;
  int blk_n = BlkM->nr_blk_cols;

  int min_blk_d = blk_m > blk_n ? blk_n : blk_m;


  res = hipMalloc(&T, BLK_SIZE_MEM);
  CHECK_CUBLAS_RETURN(res, "Failed to init T");

  for (int k = 0; k < min_blk_d; k++) {
    Numeric *A_kk = &M[BLK_POS(k, k, blk_n)];

    dgeqt2_kernel<<<1,1>>>(A_kk, T); // check res
    CHECK_ZERO_ERROR_RETURN(res, "Failed to compute dgeqt2");

    dlarfb_kernel<<<1, blk_n - k>>>(M, blk_n, k, T); // check res
    CHECK_ZERO_ERROR_RETURN(res, "Failed to compute dlarfb");

    for (int m = (k + 1); m < blk_m; m++) {

      dtsqt2_dssrfb_row_kernel<<<1, blk_m - k>>>(M, blk_n, k, m); // check res
      CHECK_ZERO_ERROR_RETURN(res, "Failed to compute row kernel");
    }
  }

  res = hipFree(&T);
  CHECK_SUCCESS_RETURN(res);

  return 0;
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// KERNEL WRAPPERS
////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void TileQR_wrapper(Numeric *A, int blk_m, int blk_n)
{
  BlockMatrix_TileQR_single_thread_kernel(A, blk_m, blk_n);
}

extern "C"
int
BlockMatrix_TileQR_single_thread(BlockMatrix *A)
{
  TileQR_wrapper<<<1, 1>>>(A->data_d, A->nr_blk_rows, A->nr_blk_cols);
  return 0;
}


__global__ void house_kernel(Numeric *x, Numeric *v, int n) {
    hipblasHandle_t handle;
    int res = hipblasCreate(&handle);
    house(&handle, x, v, n);
}

extern "C"
int
TileQR_house(hipblasHandle_t *handle, Vector *in, Vector *out) {    
    house_kernel<<<1, 1>>>(in->data_d, out->data_d, in->nr_elems);
    return 0;
}

__global__ void dgeqt2_kernel_test(Numeric *A, Numeric *T, int m, int n) {
    hipblasHandle_t handle;
    int res = hipblasCreate(&handle);

    dgeqt2(&handle, A, T, m, n);
}

extern "C"
int
TileQR_dgeqt2(hipblasHandle_t *handle, Matrix *A, Matrix *T) {    
    dgeqt2_kernel_test<<<1, 1>>>(A->data_d, T->data_d, A->nr_rows, A->nr_cols);
    return 0;
}

// Wrapper kernel to house_qr_q. Use only for testing.
__global__ void TileQR_house_qr_q_kernel(Numeric *Y,
                                         Numeric *T,
                                         Numeric *Q,
                                         Numeric *Q_,
                                         int m, int n)
{
    hipblasHandle_t handle;
    int res = hipblasCreate(&handle);

    house_qr_q(&handle, Y, T, Q, Q_, m, n);
}

// Wrapper function to single-threaded cublasDgemm_hmn kernel. Use only for testing.
extern "C"
int
TileQR_house_qr_q(Matrix *Y,
                  Matrix *T,
                  Matrix *Q,
                  Matrix *Q_,
                  int m, int n)
{    
    TileQR_house_qr_q_kernel<<<1, 1>>>(Y->data_d, T->data_d, Q->data_d, Q_->data_d, m, n);
    return 0;
}


// Wrapper kernel to cublasDgemm_hmn. Use only for testing.
__global__ void TileQR_cublasDgemm_hmn_kernel(hipblasDiagType_t diag,
                                              int m, int n, int k,
                                              const Numeric alpha,
                                              const Numeric *A, int lda,
                                              const Numeric *B, int ldb,
                                              Numeric *C, int ldc)
{
    hipblasHandle_t handle;
    int res = hipblasCreate(&handle);

    cublasDgemm_hmn(handle, diag, m, n, k, alpha, A, lda, B, ldb, C, ldc);
}

// Wrapper function to single-threaded cublasDgemm_hmn kernel. Use only for testing.
extern "C"
int
TileQR_cublasDgemm_hmn(hipblasDiagType_t diag,
                       int m, int n, int k,
                       const Numeric alpha,
                       const Numeric *A, int lda,
                       const Numeric *B, int ldb,
                       Numeric *C, int ldc)
{    
    TileQR_cublasDgemm_hmn_kernel<<<1, 1>>>(diag, m, n, k, alpha, A, lda, B, ldb, C, ldc);
    return 0;
}

// Wrapper kernel to cublasDgemm_mht. Use only for testing.
__global__ void TileQR_cublasDgemm_mht_kernel(hipblasDiagType_t diag,
                                              int m, int n, int k,
                                              const Numeric alpha,
                                              const Numeric *A, int lda,
                                              const Numeric *B, int ldb,
                                              Numeric *C, int ldc)
{
    hipblasHandle_t handle;
    int res = hipblasCreate(&handle);

    cublasDgemm_mht(handle, diag, m, n, k, alpha, A, lda, B, ldb, C, ldc);
}

// Wrapper function to single-threaded cublasDgemm_mht kernel. Use only for testing.
extern "C"
int
TileQR_cublasDgemm_mht(hipblasDiagType_t diag,
                       int m, int n, int k,
                       const Numeric alpha,
                       const Numeric *A, int lda,
                       const Numeric *B, int ldb,
                       Numeric *C, int ldc)
{    
    TileQR_cublasDgemm_mht_kernel<<<1, 1>>>(diag, m, n, k, alpha, A, lda, B, ldb, C, ldc);
    return 0;
}
