#include "hip/hip_runtime.h"
#include "BlockMatrixOperations.h"
#include "BlockMatrix.h"
#include "Vector.h"
#include "constants.h"
#include "error.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void matrixColumnSumsKernel(double *in, double *out, int nrBlkCols, double scalar)
{
	__shared__ double localColSum;

	// Each thread handles a single Block column
	int col = blockIdx.x;

	// Each thread will start at the top of a Block
	int row = BLK_LEN * threadIdx.y;

	// Index of the input
	int idx = POS(row, col, nrBlkCols);
	int idxMax = idx + BLK_LEN;

	double blockColSum = 0.0;
	for (; idx < idxMax; idx++) {
		blockColSum += in[idx];
	}

	if (row == 0) {
		localColSum = 0;
	}
	__syncthreads();
	atomicAdd(&localColSum, blockColSum);
	__syncthreads();

	if (row == 0) {
		out[col] = scalar * localColSum;
	}
}

extern "C"
int CudaBlockMatrix_cuda_column_sums(BlockMatrix *in, double *d_in, double *d_out, double scalar)
{
	dim3 dimGrid(in->nr_cols, 1);
	dim3 dimBlock(1, in->nr_blk_rows);
	
    matrixColumnSumsKernel<<<dimGrid, dimBlock>>>(d_in, d_out, in->nr_blk_cols, scalar);
    return 0;
}

// TODO: Check Cuda returns
extern "C"
int CudaBlockMatrix_column_sums(BlockMatrix *in, double *d_in, Vector *out, double scalar)
{
	int res = 0;
	double *d_out = NULL;
	const int outSize = in->nr_blk_cols * BLK_LEN * sizeof(double);

	hipMalloc((void **)&d_out, outSize);

	res = CudaBlockMatrix_cuda_column_sums(in, d_in, d_out, scalar);
	CHECK_ZERO_RETURN(res);
	
    hipMemcpy(out->data, d_out, outSize, hipMemcpyDeviceToHost);

    hipFree(d_out);
    return 0;
}

// TODO: Check Cuda returns
extern "C"
int BlockMatrix_column_sums(BlockMatrix *in, Vector *out, double scalar)
{
	double *d_in;
	int res = BlockMatrix_to_device(in, &d_in);
	CHECK_ZERO_RETURN(res);

	res = CudaBlockMatrix_column_sums(in, d_in, out, scalar);
	CHECK_ZERO_RETURN(res);

	hipFree(d_in);
	return 0;
}



