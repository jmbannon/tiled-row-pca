#include "hip/hip_runtime.h"
#include "BlockMatrixOperations.h"
#include "BlockMatrix.h"
#include "Vector.h"
#include "constants.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mykernel(void) {}

extern "C"
void someFunction(void) {
    mykernel<<<1,1>>>();
}

__global__ void matrixColumnSumsKernel(double *in, double *out, int nrBlkCols, double scalar)
{
	__shared__ double localColSum;
	// Each thread handles a single Block column
	int col = blockIdx.x;

	// Each thread will start at the top of a Block
	int row = BLK_LEN * threadIdx.y;

	// Index of the input
	int idx = POS(row, col, nrBlkCols);
	int idxMax = idx + BLK_LEN;

	double blockColSum = 0.0;
	for (; idx < idxMax; idx++) {
		blockColSum += in[idx];
	}

	atomicAdd(&localColSum, blockColSum);
	__syncthreads();

	if (row == 0) {
		out[col] = scalar * localColSum;
	}
}

extern "C"
int BlockMatrix_column_sums(BlockMatrix *in, Vector *out, double scalar)
{
	int nrBlks = in->nr_blk_rows * in->nr_blk_cols;
	double *d_in, *d_out;

	const int inSize = nrBlks * BLK_SIZE * sizeof(double);
	const int outSize = in->nr_blk_cols * BLK_LEN * sizeof(double);

	hipMalloc((void **)&d_in, inSize);
	hipMalloc((void **)&d_out, outSize);

	hipMemcpy(d_in, in->data, inSize, hipMemcpyHostToDevice);

	dim3 dimGrid(in->nr_cols, 1);
	dim3 dimBlock(1, in->nr_blk_rows);
	

    matrixColumnSumsKernel<<<dimGrid, dimBlock>>>(d_in, d_out, in->nr_blk_cols, scalar);

    hipMemcpy(out->data, d_out, outSize, hipMemcpyDeviceToHost);

    return 0;
}



