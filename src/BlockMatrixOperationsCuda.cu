#include "hip/hip_runtime.h"
#include "BlockMatrixOperations.h"
#include "BlockMatrix.h"
#include "Vector.h"
#include "constants.h"
#include "error.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void matrixColumnSumsKernel(double *in, double *out, int nrBlkCols, double scalar)
{
	__shared__ double localColSum;

	// Each thread handles a single Block column
	int col = blockIdx.x;

	// Each thread will start at the top of a Block
	int row = BLK_LEN * threadIdx.y;

	// Index of the input
	int idx = POS(row, col, nrBlkCols);
	int idxMax = idx + BLK_LEN;

	double blockColSum = 0.0;
	for (; idx < idxMax; idx++) {
		blockColSum += in[idx];
	}

	if (row == 0) {
		localColSum = 0;
	}
	__syncthreads();
	atomicAdd(&localColSum, blockColSum);
	__syncthreads();

	if (row == 0) {
		out[col] = scalar * localColSum;
	}
}

extern "C"
int CudaBlockMatrix_cuda_column_sums(BlockMatrix *in, double *d_out, double scalar)
{
	dim3 dimGrid(in->nr_cols, 1);
	dim3 dimBlock(1, in->nr_blk_rows);
	
    matrixColumnSumsKernel<<<dimGrid, dimBlock>>>(in->data_d, d_out, in->nr_blk_cols, scalar);
    return 0;
}

// TODO: Check Cuda returns
extern "C"
int BlockMatrix_device_column_sums(BlockMatrix *in, Vector *out, double scalar)
{
	int res = 0;
	double *d_out = NULL;
	const int outSize = in->nr_blk_cols * BLK_LEN * sizeof(double);

	res = hipMalloc((void **)&d_out, outSize);
    CHECK_SUCCESS_RETURN(res);

	res = CudaBlockMatrix_cuda_column_sums(in, d_out, scalar);
	CHECK_ZERO_RETURN(res);
	
    res = hipMemcpy(out->data, d_out, outSize, hipMemcpyDeviceToHost);
    CHECK_SUCCESS_RETURN(res);

    res = hipFree(d_out);
    CHECK_SUCCESS_RETURN(res);
    
    return 0;
}


