#include "Matrix.h"
#include "constants.h"
#include "error.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

/**
 * Copies data from host to device.
 */
extern "C"
int
Matrix_copy_host_to_device(Matrix *in)
{
    int res = hipMemcpy(in->data_d, in->data, Matrix_size_bytes(in), hipMemcpyHostToDevice);
    CHECK_SUCCESS_RETURN(res);

    return 0;
}

extern "C"
int
Matrix_copy_device_to_host(Matrix *in)
{
    int res = hipMemcpy(in->data, in->data_d, Matrix_size_bytes(in), hipMemcpyDeviceToHost);
    CHECK_SUCCESS_RETURN(res);

    return 0;
}

/**
 * CudaMalloc device matrix.
 */
extern "C"
int
Matrix_init_device(Matrix *in)
{
    int res = hipMalloc((void **)&in->data_d, Matrix_size_bytes(in));
    CHECK_SUCCESS_RETURN(res);

    return 0;
}

extern "C"
int
Matrix_free_device(Matrix *in)
{
    int res = hipFree(in->data_d);
    CHECK_SUCCESS_RETURN(res);

    return 0;
}