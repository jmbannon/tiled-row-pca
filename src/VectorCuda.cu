#include "Vector.h"
#include "error.h"
#include "constants.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

extern "C"
int
Vector_copy_host_to_device(Vector *in)
{
    int res = hipMemcpy(in->data_d, in->data, Vector_size_bytes(in), hipMemcpyHostToDevice);
    CHECK_SUCCESS_RETURN(res);

    return 0;
}

extern "C"
int
Vector_copy_device_to_host(Vector *in)
{
    int res = hipMemcpy(in->data, in->data_d, Vector_size_bytes(in), hipMemcpyDeviceToHost);
    CHECK_SUCCESS_RETURN(res);

    return 0;
}

extern "C"
int
Vector_init_device(Vector *in)
{
    int res = hipMalloc((void **)&in->data_d, Vector_size_bytes(in));
    CHECK_SUCCESS_RETURN(res);

    return 0;
}

extern "C"
int
Vector_free_device(Vector *in)
{
    int res = hipFree(in->data_d);
    CHECK_SUCCESS_RETURN(res);

    return 0;
}