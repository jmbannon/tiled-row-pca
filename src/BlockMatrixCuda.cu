#include "BlockMatrix.h"
#include "constants.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// TODO: check return values of CUDA functions
extern "C"
int
BlockMatrix_to_device(BlockMatrix *in, double **d_in)
{
    int nrBlks = in->nr_blk_rows * in->nr_blk_cols;

    const int inSize = nrBlks * BLK_SIZE * sizeof(double);
    hipMalloc((void **)d_in, inSize);

    hipMemcpy(*d_in, in->data, inSize, hipMemcpyHostToDevice);

    return 0;
}